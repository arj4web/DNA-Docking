#include "hip/hip_runtime.h"
/*
This file is part of ftdock, a program for rigid-body protein-protein docking 
Copyright (C) 1997-2000 Gidon Moont

Biomolecular Modelling Laboratory
Imperial Cancer Research Fund
44 Lincoln's Inn Fields
London WC2A 3PX

+44 (0)20 7269 3348
http://www.bmm.icnet.uk/

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.

*/

#include "structures.cuh"

__global__ void zero1_interaction_grid(hipfftReal *grid,int grid_size)
{
    int x=threadIdx.x+(blockDim.x*blockIdx.x);
    int y=threadIdx.y+(blockDim.y*blockIdx.y);
    int z=threadIdx.z+(blockDim.z*blockIdx.z);

    if(z<grid_size&&x<grid_size&&y<grid_size)grid[gaddress(x,y,z,grid_size)] = (hipfftReal)0;
}

__global__ void interaction_grid(hipfftReal *grid, Amino_Acid *Residue,float grid_span , int grid_size ,int steps,int ydim)
{
  int residue=threadIdx.y+(blockDim.y*blockIdx.y);
  int atom=threadIdx.x+(blockDim.x*blockIdx.x);
    int x_step , y_step , z_step ;

     float		x_centre , y_centre , z_centre ;

  /* Variables */

     float         distance,one_span;
     one_span = grid_span / (float)grid_size ;

     distance = 1.8 ;

if(residue<ydim){

    if((residue>0)&&(atom>0)&&(atom<=Residue[residue].size))
    {

        
        int x = gord(Residue[residue].Atom[atom].coord[1] , grid_span , grid_size );
        int y = gord(Residue[residue].Atom[atom].coord[2] , grid_span , grid_size );
        int z = gord(Residue[residue].Atom[atom].coord[3] , grid_span , grid_size );

        for( x_step = max( ( x - steps ) , 0 ) ; x_step <= min( ( x + steps ) , ( grid_size - 1 ) ) ; x_step ++ ) {

            x_centre  = gcentre( x_step , grid_span , grid_size ) ;

        for( y_step = max( ( y - steps ) , 0 ) ; y_step <= min( ( y + steps ) , ( grid_size - 1 ) ) ; y_step ++ ) {

          y_centre  = gcentre( y_step , grid_span , grid_size ) ;

          for( z_step = max( ( z - steps ) , 0 ) ; z_step <= min( ( z + steps ) , ( grid_size - 1 ) ) ; z_step ++ ) {

            z_centre  = gcentre( z_step , grid_span , grid_size ) ;

            if( pythagoras(Residue[residue].Atom[atom].coord[1] ,Residue[residue].Atom[atom].coord[2] ,Residue[residue].Atom[atom].coord[3] , x_centre , y_centre , z_centre ) < distance ) grid[gaddress(x_step,y_step,z_step,grid_size)] = (hipfftReal)1 ;

          }
        }
     


    }

    }
  }
}

void discretise_structure( struct Structure This_Structure , float grid_span , int grid_size , hipfftReal *grid, int size1 ) {

/************/

  /* Counters */

 

  /* Co-ordinates */


  int	steps;


  /* Variables */

  float         distance , one_span ;

/************/

  one_span = grid_span / (float)grid_size ;

  distance = 1.8 ;

/************/
dim3 numblocks(((grid_size-1)/threadperblock3D.x)+1,((grid_size-1)/threadperblock3D.y)+1,((grid_size-1)/threadperblock3D.z)+1);



zero1_interaction_grid<<<numblocks,threadperblock3D>>>(grid,grid_size);
hipDeviceSynchronize();


/************/
struct Amino_Acid *Residue,*d_Residue;
Residue = (struct Amino_Acid*)malloc((This_Structure.length+1)*sizeof(Amino_Acid));
int a=0;
for (int i = 1; i <=This_Structure.length; i++)
{
  Residue[i]=This_Structure.Residue[i];
  hipMalloc(&Residue[i].Atom,(This_Structure.Residue[i].size+1)*sizeof(struct Atom));
  hipMemcpy(Residue[i].Atom,This_Structure.Residue[i].Atom,(This_Structure.Residue[i].size+1)*sizeof(struct Atom),hipMemcpyHostToDevice);
  a=max(a,This_Structure.Residue[i].size);
  
}
hipMalloc((void**)&d_Residue,(This_Structure.length+1)*sizeof(struct Amino_Acid));
hipMemcpy(d_Residue,Residue,(This_Structure.length+1)*sizeof(struct Amino_Acid),hipMemcpyHostToDevice);

  dim3 numblock1((a/threadperblock2D.x)+1,(This_Structure.length/threadperblock2D.y)+1);
  steps = (int)( ( distance / one_span ) + 1.5 ) ;
  interaction_grid<<<numblock1,threadperblock2D>>>(grid, d_Residue, grid_span,grid_size,steps,This_Structure.length+1);
  hipDeviceSynchronize();
  hipFree(d_Residue);
  
  free(Residue);
  /************/

  return ;

}



/************************/




__global__ void surface_grid( float grid_span , int grid_size , hipfftReal *grid , float surface , float internal_value ) {


/************/

  /* Counters */

  int	x=threadIdx.x+(blockIdx.x*blockDim.x) , y=threadIdx.y+(blockIdx.y*blockDim.y), z=threadIdx.z +(blockIdx.z*blockDim.z);
  int	steps , x_step , y_step , z_step ;

  /* Variables */

  float		one_span ;

  int	at_surface ;

/************/
if(z<grid_size&&x<grid_size&&y<grid_size){

  one_span = grid_span / (float)grid_size ;

/************/

  /* Surface grid atoms */

  steps = (int)( ( surface / one_span ) + 1.5 ) ;

  
        if( (int)grid[gaddress(x,y,z,grid_size)] == 1 ) {

          at_surface = 0 ;

          for( x_step = max( x - steps , 0 ) ; x_step <= min( x + steps , grid_size - 1 ) ; x_step ++ ) {
            for( y_step = max( y - steps , 0 ) ; y_step <= min( y + steps , grid_size - 1 ) ; y_step ++ ) {
              for( z_step = max( z - steps , 0 ) ; z_step <= min( z + steps , grid_size - 1 ) ; z_step ++ ) {

                if( (int)grid[gaddress(x_step,y_step,z_step,grid_size)] == 0 ) {

                  if( ( (float)( ( ( x_step - x ) * ( x_step - x ) ) + ( ( y_step - y ) * ( y_step - y ) ) + ( ( z_step - z ) * ( z_step - z ) ) ) * one_span * one_span ) < ( surface * surface ) ) at_surface = 1 ;

                }

              }
            }
          }

          if( at_surface == 0 ) grid[gaddress(x,y,z,grid_size)] = (hipfftReal)internal_value ;

        }
}
/************/

  return ;

}
